#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include "cuda-utils.h"
#include "utils.h"

void allocateArraysDevice(std::vector<double*> arrays, uint32 size) {

	for (unsigned i = 0; i < arrays.size(); i++) {
		HANDLE_ERROR(hipMalloc((void** ) &arrays[i], size * sizeof(double)));
	}

}

__global__ void correctionCycle(double* surfaceTemperatureLine, double* zomLine,
		double* ustarRLine, double* ustarWLine, double* rahRLine,
		double* rahWLine, double *a, double *b, double *u200) {

	//Identify position
	int pos = 0;

	double sensibleHeatFlux = RHO * SPECIFIC_HEAT_AIR
			* (*a + *b * (surfaceTemperatureLine[pos] - 273.15)) / rahRLine[pos];

	double ustarPow3 = ustarRLine[pos] * ustarRLine[pos] * ustarRLine[pos];

	double L = -1
			* ((RHO * SPECIFIC_HEAT_AIR * ustarPow3
					* surfaceTemperatureLine[pos])
					/ (VON_KARMAN * GRAVITY * sensibleHeatFlux));

	double y01 = pow((1 - (16 * 0.1) / L), 0.25);
	double y2 = pow((1 - (16 * 2) / L), 0.25);
	double x200 = pow((1 - (16 * 200) / L), 0.25);

	double psi01, psi2, psi200;

	if (!isnan(L) && L > 0) {

		psi01 = -5 * (0.1 / L);
		psi2 = -5 * (2 / L);
		psi200 = -5 * (2 / L);

	} else {

		psi01 = 2 * log((1 + y01 * y01) / 2);

		psi2 = 2 * log((1 + y2 * y2) / 2);

		psi200 = 2 * log((1 + x200) / 2) + log((1 + x200 * x200) / 2)
				- 2 * atan(x200) + 0.5 * M_PI;

	}

	ustarWLine[pos] = (VON_KARMAN * *u200) / (log(200 / zomLine[pos]) - psi200);

	rahWLine[pos] = (log(2 / 0.1) - psi2 + psi01)
			/ (ustarWLine[pos] * VON_KARMAN);

}

int main(int argc, char **argv) {

	/*********** FIRST ZOM, USTAR E RAH VALUES BEGIN **********/

	std::string albedoPath, zomPath, ustarPath0, ustarPath1, rahPath0, rahPath1,
			sensibleHeatPath, surfaceTemperaturePath; //TODO insert path

	//Albedo base TIFF
	TIFF *albedo;
	albedo = TIFFOpen(albedoPath.c_str(), "rm");

	uint32 heightBand, widthBand;
	TIFFGetField(albedo, TIFFTAG_IMAGELENGTH, &heightBand);
	TIFFGetField(albedo, TIFFTAG_IMAGEWIDTH, &widthBand);

	//Auxiliary products TIFFs
	TIFF *zom, *ustar, *aerodynamicResistance;
	zom = TIFFOpen(zomPath.c_str(), "w8m");
	setup(zom, albedo);

	ustar = TIFFOpen(ustarPath0.c_str(), "w8m");
	setup(ustar, albedo);

	aerodynamicResistance = TIFFOpen(rahPath0.c_str(), "w8m");
	setup(aerodynamicResistance, albedo);

	//Calculates initial values of zom, ustar and aerodynamic_resistance TODO

	TIFFClose(albedo);
	TIFFClose(zom);
	TIFFClose(ustar);
	TIFFClose(aerodynamicResistance);

	/*********** FIRST ZOM, USTAR E RAH VALUES END **********/

	/*********** DEALING WITH HOT PIXEL BEGIN **********/

	aerodynamicResistance = TIFFOpen(rahPath0.c_str(), "rm");

	//Extract the hot pixel aerodynamic_resistance //TODO deal with hot pixel
	//hot_pixel.aerodynamic_resistance.push_back(read_position_tiff(aerodynamic_resistance, hot_pixel.col, hot_pixel.line));
	//double H_hot = hot_pixel.net_radiation - hot_pixel.soil_heat_flux;
	double hHot = 154564;

	TIFFClose(aerodynamicResistance);

	/*********** DEALING WITH HOT PIXEL END **********/

	/********** RAH CYCLE BEGIN **********/

	TIFF *ustarR, *aerodynamicResistanceR;
	TIFF *ustarW, *aerodynamicResistanceW, *sensibleHeatFlux;
	TIFF *surfaceTemperature;
	zom = TIFFOpen(zomPath.c_str(), "rm"); //It's not modified into the rah cycle
	surfaceTemperature = TIFFOpen(surfaceTemperaturePath.c_str(), "rm");

	//It's only written into the rah cycle
	sensibleHeatFlux = TIFFOpen(sensibleHeatPath.c_str(), "w8m");
	setup(sensibleHeatFlux, zom);

	//Auxiliaries arrays calculation
	double zomLine[widthBand], surfaceTemperatureLine[widthBand];
	double ustarReadLine[widthBand], ustarWriteLine[widthBand];
	double aerodynamicResistanceReadLine[widthBand],
			aerodynamicResistanceWriteLine[widthBand];

	//Auxiliaries arrays calculation to device
	double *devZom, *devTS;
	double *devUstarR, *devUstarW;
	double *devRahR, *devRahW;

	//Allocating arrays on device
	allocateArraysDevice(std::vector<double*> { devZom, devTS, devUstarR, devUstarW,
			devRahR, devRahW }, widthBand);

	//Auxiliaries loop variables
	int i = 0;
	bool Erro = true;
	double rahHot0, rahHot;

	while (Erro) {

		rahHot0 = 1245; //TODO hotPixel.aerodynamicResistance[i];

		//Opening the ustar e rah TIFFs for read and write based on i parity
		//If i is even then the TIFFs with 0 in the path name will be readable
		//otherwise TIFFs with 1 will be.

		if (i % 2) {

			//Since ustar is both write and read into the rah cycle, two TIFF will be needed
			ustarR = TIFFOpen(ustarPath0.c_str(), "rm");
			ustarW = TIFFOpen(ustarPath1.c_str(), "w8m");
			setup(ustarW, zom);

			//Since ustar is both write and read into the rah cycle, two TIFF will be needed
			aerodynamicResistanceR = TIFFOpen(rahPath0.c_str(), "rm");
			aerodynamicResistanceW = TIFFOpen(rahPath1.c_str(), "w8m");
			setup(aerodynamicResistanceW, zom);

		} else {

			//Since ustar is both write and read into the rah cycle, two TIFF will be needed
			ustarR = TIFFOpen(ustarPath1.c_str(), "rm");
			ustarW = TIFFOpen(ustarPath0.c_str(), "w8m");
			setup(ustarW, zom);

			//Since ustar is both write and read into the rah cycle, two TIFF will be needed
			aerodynamicResistanceR = TIFFOpen(rahPath1.c_str(), "rm");
			aerodynamicResistanceW = TIFFOpen(rahPath0.c_str(), "w8m");
			setup(aerodynamicResistanceW, zom);

		}

		//Coefficients calculation
//		double dtHot = hHot * rahHot0 / (RHO * SPECIFIC_HEAT_AIR);
//		double b = dtHot / (hotPixel.temperature - coldPixel.temperature);
//		double a = -b * (coldPixel.temperature - 273.15); TODO deal with hot/cold pixel

		double dtHot = hHot * rahHot0 / (RHO * SPECIFIC_HEAT_AIR);
		double b = 154;
		double a = 7864;

		double u200 = 35.654654; //TODO

		double *devA, *devB, *devU200;

		/********** COPY HOST TO DEVICE MEMORY BEGIN TODO **********/

		HANDLE_ERROR(hipMemcpy(devA, (void**) &a, sizeof(int), hipMemcpyHostToDevice));

		HANDLE_ERROR(hipMemcpy(devB, (void**) &b, sizeof(int), hipMemcpyHostToDevice));

		HANDLE_ERROR(
				hipMemcpy(devU200, (void**) &u200, sizeof(double),
						hipMemcpyHostToDevice));

		/********** COPY HOST TO DEVICE MEMORY END **********/

		for (int line = 0; line < heightBand; line++) {

			//Reading data needed
			read_line_tiff(surfaceTemperature, surfaceTemperatureLine, line);
			read_line_tiff(zom, zomLine, line);
			read_line_tiff(ustarR, ustarReadLine, line);
			read_line_tiff(aerodynamicResistanceR,
					aerodynamicResistanceReadLine, line);

			/********** COPY HOST TO DEVICE MEMORY BEGIN **********/

			HANDLE_ERROR(
					hipMemcpy(devTS, surfaceTemperatureLine,
							widthBand * sizeof(double),
							hipMemcpyHostToDevice));

			HANDLE_ERROR(
					hipMemcpy(devZom, zomLine, widthBand * sizeof(double),
							hipMemcpyHostToDevice));

			HANDLE_ERROR(
					hipMemcpy(devUstarR, ustarReadLine,
							widthBand * sizeof(double),
							hipMemcpyHostToDevice));

			HANDLE_ERROR(
					hipMemcpy(devRahR, aerodynamicResistanceReadLine,
							widthBand * sizeof(double),
							hipMemcpyHostToDevice));

			/********** COPY HOST TO DEVICE MEMORY END **********/

			/********** KERNEL BEGIN **********/

			correctionCycle<<<10, 10>>>(devTS, devZom, devUstarR, devUstarW, devRahR, devRahW, devA, devB, devU200);

			/********** KERNEL END **********/

			/********** COPY DEVICE TO HOST MEMORY BEGIN **********/

			HANDLE_ERROR(
					hipMemcpy(ustarWriteLine, devUstarW,
							widthBand * sizeof(double),
							hipMemcpyDeviceToHost));

			HANDLE_ERROR(
					hipMemcpy(aerodynamicResistanceWriteLine, devRahW,
							widthBand * sizeof(double),
							hipMemcpyDeviceToHost));

			/********** COPY DEVICE TO HOST MEMORY END **********/

			save_tiffs(std::vector<double*> { ustarWriteLine,
					aerodynamicResistanceWriteLine }, std::vector<TIFF*> { ustarW,
					aerodynamicResistanceW }, line);

		}

	}

	/********** RAH CYCLE END **********/

	return 0;
}
